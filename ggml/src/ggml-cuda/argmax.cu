#include "hip/hip_runtime.h"
#include "common.cuh"
#include "argmax.cuh"
#include "sum.cuh"

#include <cstdint>

static __global__ void argmax_f32(
    const float * x, int32_t * dst, const int64_t ncols, const int64_t nrows) {

    int argmax_thread = 0;
    const int64_t row0 = (int64_t)blockIdx.x*WARP_SIZE;

#pragma unroll
    for (int64_t row1 = 0; row1 < WARP_SIZE; ++row1) {
        const int64_t row = row0 + row1;

        if (row >= nrows) {
            break;
        }

        float maxval = -FLT_MAX;
        int   argmax = -1;

        for (int32_t col = threadIdx.x; col < ncols; col += WARP_SIZE) {
            const float val        = x[row*ncols + col];
            const int   bigger     = val > maxval;
            const int   not_bigger = bigger ^ 0x00000001;

            maxval = maxval*not_bigger + val*bigger;
            argmax = argmax*not_bigger + col*bigger;
        }

#pragma unroll
        for (int mask = 16; mask > 0; mask >>= 1) {
            const float val        = __shfl_xor_sync(0xFFFFFFFF, maxval, mask, WARP_SIZE);
            const int   col        = __shfl_xor_sync(0xFFFFFFFF, argmax, mask, WARP_SIZE);
            const int   bigger     = val > maxval;
            const int   not_bigger = bigger ^ 0x00000001;

            maxval = maxval*not_bigger + val*bigger;
            argmax = argmax*not_bigger + col*bigger;
        }

        const int store = row1 == threadIdx.x;
        argmax_thread += store*argmax;
    }

    const int row = row0 + threadIdx.x;

    if (row >= nrows) {
        return;
    }

    dst[row] = argmax_thread;
}

void ggml_cuda_argmax(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);

    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ne00  = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    const float * src0_d = (const float *) src0->data;
    int32_t     * dst_d  = (int32_t     *) dst->data;

    hipStream_t stream = ctx.stream();

    const int64_t num_blocks = (nrows + WARP_SIZE - 1) / WARP_SIZE;

    const dim3 blocks_dim(WARP_SIZE, 1, 1);
    const dim3 blocks_num(num_blocks, 1, 1);

    argmax_f32<<<blocks_num, blocks_dim, 0, stream>>>(src0_d, dst_d, ne00, nrows);
}
