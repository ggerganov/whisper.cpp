#include "hip/hip_runtime.h"
#include "unary.cuh"

template <class T>
static __global__ void op_abs(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = fabsf(x[i]);
}

template <class T>
static __global__ void op_sgn(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)(x[i] > (T)0.f ? 1.f : ((x[i] < (T)0.f ? -1.f : 0.f)));
}

template <class T>
static __global__ void op_neg(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = -x[i];
}

template <class T>
static __global__ void op_step(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = x[i] > (T)0.0f;
}

template <class T>
static __global__ void op_gelu(const T * x, T * dst, const int k) {
    const T GELU_COEF_A    = 0.044715f;
    const T SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    T xi = x[i];
    dst[i] = (T)0.5f*xi*((T)1.0f + (T)tanhf(SQRT_2_OVER_PI*xi*((T)1.0f + GELU_COEF_A*xi*xi)));
}

template <class T>
static __global__ void op_gelu_quick(const T * x, T * dst, int k) {
    const T GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * ((T)1.0f / ((T)1.0f + (T)expf(GELU_QUICK_COEF * x[i])));
}

template <class T>
static __global__ void op_silu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void op_silu_back(
        const T * grad, const T * xf, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const T xfi = xf[i];
    const T s = (T)1.0f / ((T)1.0f + (T)expf(-xfi));
    dst[i] = grad[i] * s * ((T)1.0f + xfi * ((T)1.0f - s));
}

template <class T>
static __global__ void op_tanh(const T * x, T * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

template <class T>
static __global__ void op_relu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

template <class T>
static __global__ void op_sigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = (T)1.0f / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void op_hardsigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void op_hardswish(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (T)fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void op_exp(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = expf(x[i]);
}

template <class T>
static __global__ void op_leaky_relu(const T * x, T * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = (T)fmaxf(x[i], 0) + (T)fminf(x[i], 0.0f) * (T)negative_slope;
}

template <class T>
static __global__ void op_sqr(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

template <class T>
static __global__ void op_sqrt(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

template <class T>
static __global__ void op_sin(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sinf(x[i]);
}

template <class T>
static __global__ void op_cos(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = cosf(x[i]);
}

template <class T>
static __global__ void op_log(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = logf(x[i]);
}

template <class T>
static void abs_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_abs<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sgn_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_sgn<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void neg_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    op_neg<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void step_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_STEP_BLOCK_SIZE - 1) / CUDA_STEP_BLOCK_SIZE;
    op_step<<<num_blocks, CUDA_STEP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    op_gelu<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_quick_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    op_gelu_quick<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    op_silu<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_back_cuda(const T * grad, const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BACK_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    op_silu_back<<<num_blocks, CUDA_SILU_BACK_BLOCK_SIZE, 0, stream>>>(grad, x, dst, k);
}

template <class T>
static void tanh_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    op_tanh<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void relu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    op_relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    op_sigmoid<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardsigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    op_hardsigmoid<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardswish_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    op_hardswish<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void exp_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_EXP_BLOCK_SIZE - 1) / CUDA_EXP_BLOCK_SIZE;
    op_exp<<<num_blocks, CUDA_EXP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void leaky_relu_cuda(const T * x, T * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    op_leaky_relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

template <class T>
static void sqr_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    op_sqr<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sqrt_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    op_sqrt<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sin_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIN_BLOCK_SIZE - 1) / CUDA_SIN_BLOCK_SIZE;
    op_sin<<<num_blocks, CUDA_SIN_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void cos_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    op_cos<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void log_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    op_log<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

void ggml_cuda_op_abs(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        abs_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        abs_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sgn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sgn_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sgn_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        neg_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        neg_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        step_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        step_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // input from forward pass
    const ggml_tensor * src1 = dst->src[1]; // grads of forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_back_cuda((const half *)src0_d, (const half *)src1_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_back_cuda((const float*)src0_d, (const float*)src1_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_quick_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_quick_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        tanh_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        tanh_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardsigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardsigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardswish_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardswish_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        exp_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        exp_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        leaky_relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), negative_slope, stream);
    } else {
        leaky_relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), negative_slope, stream);
    }
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqr_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqr_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqrt_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqrt_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sin_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sin_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        cos_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        cos_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_log(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        log_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        log_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}
