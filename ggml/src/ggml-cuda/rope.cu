#include "hip/hip_runtime.h"
#include "rope.cuh"

struct rope_corr_dims {
    float v[2];
};

static __device__ float rope_yarn_ramp(const float low, const float high, const int i0) {
    const float y = (i0 / 2 - low) / max(0.001f, high - low);
    return 1.0f - min(1.0f, max(0.0f, y));
}

// YaRN algorithm based on LlamaYaRNScaledRotaryEmbedding.py from https://github.com/jquesnelle/yarn
// MIT licensed. Copyright (c) 2023 Jeffrey Quesnelle and Bowen Peng.
static __device__ void rope_yarn(
    float theta_extrap, float freq_scale, rope_corr_dims corr_dims, int64_t i0, float ext_factor, float mscale,
    float * cos_theta, float * sin_theta) {
    // Get n-d rotational scaling corrected for extrapolation
    float theta_interp = freq_scale * theta_extrap;
    float theta = theta_interp;
    if (ext_factor != 0.0f) {
        float ramp_mix = rope_yarn_ramp(corr_dims.v[0], corr_dims.v[1], i0) * ext_factor;
        theta = theta_interp * (1 - ramp_mix) + theta_extrap * ramp_mix;

        // Get n-d magnitude scaling corrected for interpolation
        mscale *= 1.0f + 0.1f * logf(1.0f / freq_scale);
    }
    *cos_theta = cosf(theta) * mscale;
    *sin_theta = sinf(theta) * mscale;
}

template<typename T, bool has_ff>
static __global__ void rope_norm(
    const T * x, T * dst, int ne0, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ne0 + i0;
    const int i2 = row/p_delta_rows;

    const float theta_base = pos[i2]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + 1];

    dst[i + 0] = x0*cos_theta - x1*sin_theta;
    dst[i + 1] = x0*sin_theta + x1*cos_theta;
}

template<typename T, bool has_ff>
static __global__ void rope_neox(
    const T * x, T * dst, int ne0, int n_dims, const int32_t * pos, float freq_scale, int p_delta_rows,
    float ext_factor, float attn_factor, rope_corr_dims corr_dims, float theta_scale, const float * freq_factors) {
    const int i0 = 2*(blockDim.y*blockIdx.y + threadIdx.y);

    if (i0 >= ne0) {
        return;
    }

    const int row = blockDim.x*blockIdx.x + threadIdx.x;

    if (i0 >= n_dims) {
        const int i = row*ne0 + i0;

        dst[i + 0] = x[i + 0];
        dst[i + 1] = x[i + 1];

        return;
    }

    const int i  = row*ne0 + i0/2;
    const int i2 = row/p_delta_rows;

    const float theta_base = pos[i2]*powf(theta_scale, i0/2.0f);

    const float freq_factor = has_ff ? freq_factors[i0/2] : 1.0f;

    float cos_theta;
    float sin_theta;

    rope_yarn(theta_base/freq_factor, freq_scale, corr_dims, i0, ext_factor, attn_factor, &cos_theta, &sin_theta);

    const float x0 = x[i + 0];
    const float x1 = x[i + n_dims/2];

    dst[i + 0]        = x0*cos_theta - x1*sin_theta;
    dst[i + n_dims/2] = x0*sin_theta + x1*cos_theta;
}

template<typename T>
static void rope_norm_cuda(
    const T * x, T * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_norm<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    } else {
        rope_norm<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    }
}

template<typename T>
static void rope_neox_cuda(
    const T * x, T * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {
    GGML_ASSERT(ne0 % 2 == 0);
    const dim3 block_dims(1, CUDA_ROPE_BLOCK_SIZE, 1);
    const int n_blocks_x = (ne0 + 2*CUDA_ROPE_BLOCK_SIZE - 1) / (2*CUDA_ROPE_BLOCK_SIZE);
    const dim3 block_nums(nr, n_blocks_x, 1);

    const float theta_scale = powf(freq_base, -2.0f/n_dims);

    if (freq_factors == nullptr) {
        rope_neox<T, false><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    } else {
        rope_neox<T, true><<<block_nums, block_dims, 0, stream>>>(
                x, dst, ne0, n_dims, pos, freq_scale, p_delta_rows, ext_factor, attn_factor, corr_dims,
                theta_scale, freq_factors
                );
    }
}

static void rope_norm_cuda_f16(
    const half * x, half * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_norm_cuda<half>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_norm_cuda_f32(
    const float * x, float * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_norm_cuda<float>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_neox_cuda_f16(
    const half * x, half * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream) {

    rope_neox_cuda<half>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

static void rope_neox_cuda_f32(
    const float * x, float * dst, int ne0, int n_dims, int nr, const int32_t * pos, float freq_scale, int p_delta_rows,
    float freq_base, float ext_factor, float attn_factor, rope_corr_dims corr_dims, const float * freq_factors, hipStream_t stream
) {

    rope_neox_cuda<float>(x, dst, ne0, n_dims, nr, pos, freq_scale, p_delta_rows, freq_base, ext_factor, attn_factor, corr_dims, freq_factors, stream);
}

void ggml_cuda_op_rope(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const ggml_tensor * src2 = dst->src[2];

    const float * src0_d = (const float *)src0->data;
    const float * src1_d = (const float *)src1->data;

    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t nr = ggml_nrows(src0);

    //const int n_past     = ((int32_t *) dst->op_params)[0];
    const int n_dims     = ((int32_t *) dst->op_params)[1];
    const int mode       = ((int32_t *) dst->op_params)[2];
    //const int n_ctx      = ((int32_t *) dst->op_params)[3];
    const int n_ctx_orig = ((int32_t *) dst->op_params)[4];

    // RoPE alteration for extended context
    float freq_base;
    float freq_scale;
    float ext_factor;
    float attn_factor;
    float beta_fast;
    float beta_slow;

    memcpy(&freq_base,   (int32_t *) dst->op_params +  5, sizeof(float));
    memcpy(&freq_scale,  (int32_t *) dst->op_params +  6, sizeof(float));
    memcpy(&ext_factor,  (int32_t *) dst->op_params +  7, sizeof(float));
    memcpy(&attn_factor, (int32_t *) dst->op_params +  8, sizeof(float));
    memcpy(&beta_fast,   (int32_t *) dst->op_params +  9, sizeof(float));
    memcpy(&beta_slow,   (int32_t *) dst->op_params + 10, sizeof(float));

    const bool is_neox = mode & GGML_ROPE_TYPE_NEOX;

    const int32_t * pos = (const int32_t *) src1_d;

    const float * freq_factors = nullptr;
    if (src2 != nullptr) {
        freq_factors = (const float *) src2->data;
    }

    rope_corr_dims corr_dims;
    ggml_rope_yarn_corr_dims(n_dims, n_ctx_orig, freq_base, beta_fast, beta_slow, corr_dims.v);

    // compute
    if (is_neox) {
        if (src0->type == GGML_TYPE_F32) {
            rope_neox_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_neox_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    } else {
        if (src0->type == GGML_TYPE_F32) {
            rope_norm_cuda_f32(
                (const float *)src0_d, (float *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else if (src0->type == GGML_TYPE_F16) {
            rope_norm_cuda_f16(
                (const half *)src0_d, (half *)dst_d, ne00, n_dims, nr, pos, freq_scale, ne01, freq_base, ext_factor,
                attn_factor, corr_dims, freq_factors, stream
            );
        } else {
            GGML_ABORT("fatal error");
        }
    }
}
