#include "hip/hip_runtime.h"
#include "dmmv.cuh"
#include "dequantize.cuh"
#include "convert.cuh"

#ifndef K_QUANTS_PER_ITERATION
#define K_QUANTS_PER_ITERATION 2
#else
static_assert(K_QUANTS_PER_ITERATION == 1 || K_QUANTS_PER_ITERATION == 2, "K_QUANTS_PER_ITERATION must be 1 or 2");
#endif

static __global__ void dequantize_mul_mat_vec_q2_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q2_K * x = (const block_q2_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...15
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 16/K_QUANTS_PER_ITERATION;

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15 or 0...14 in steps of 2
    const int q_offset = 32*im + l0;
    const int s_offset = 8*im;
    const int y_offset = 128*im + l0;

    uint32_t aux[4];
    const uint8_t * d = (const uint8_t *)aux;
    const uint8_t * m = (const uint8_t *)(aux + 2);

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint32_t * a = (const uint32_t *)(x[i].scales + s_offset);
        aux[0] = a[0] & 0x0f0f0f0f;
        aux[1] = a[1] & 0x0f0f0f0f;
        aux[2] = (a[0] >> 4) & 0x0f0f0f0f;
        aux[3] = (a[1] >> 4) & 0x0f0f0f0f;

        float sum1 = 0, sum2 = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            sum1 += y[l+ 0] * d[0] * ((q[l+ 0] >> 0) & 3)
                  + y[l+32] * d[2] * ((q[l+ 0] >> 2) & 3)
                  + y[l+64] * d[4] * ((q[l+ 0] >> 4) & 3)
                  + y[l+96] * d[6] * ((q[l+ 0] >> 6) & 3)
                  + y[l+16] * d[1] * ((q[l+16] >> 0) & 3)
                  + y[l+48] * d[3] * ((q[l+16] >> 2) & 3)
                  + y[l+80] * d[5] * ((q[l+16] >> 4) & 3)
                  +y[l+112] * d[7] * ((q[l+16] >> 6) & 3);
            sum2 += y[l+ 0] * m[0] + y[l+32] * m[2] + y[l+64] * m[4] + y[ l+96] * m[6]
                  + y[l+16] * m[1] + y[l+48] * m[3] + y[l+80] * m[5] + y[l+112] * m[7];

        }
        tmp += dall * sum1 - dmin * sum2;

    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q3_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q3_K * x = (const block_q3_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const uint16_t kmask1 = 0x0303;
    const uint16_t kmask2 = 0x0f0f;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int n  = K_QUANTS_PER_ITERATION;               // iterations in the inner loop
    const int step = 16/K_QUANTS_PER_ITERATION;
    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0....15 or 0...7

    const uint8_t m = 1 << (4*im);

    const int l0 = n*in;                                 // 0...15 or 0...14 in steps of 2
    const int q_offset =  32*im + l0;
    const int y_offset = 128*im + l0;

    uint16_t utmp[4];
    const int8_t * s = (const int8_t *)utmp;

    const uint16_t s_shift = 4*im;

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;
        const uint8_t * h = x[i].hmask + l0;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        utmp[0] = ((a[0] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 0)) & kmask1) << 4);
        utmp[1] = ((a[1] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 0)) & kmask1) << 4);
        utmp[2] = ((a[2] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 2)) & kmask1) << 4);
        utmp[3] = ((a[3] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 2)) & kmask1) << 4);

        const float d = x[i].d;

        float sum = 0;
        for (int l = 0; l < n; ++l) {
            sum += y[l+ 0] * (s[0] - 32) * (((q[l] >> 0) & 3) - (h[l] & (m << 0) ? 0 : 4))
                 + y[l+32] * (s[2] - 32) * (((q[l] >> 2) & 3) - (h[l] & (m << 1) ? 0 : 4))
                 + y[l+64] * (s[4] - 32) * (((q[l] >> 4) & 3) - (h[l] & (m << 2) ? 0 : 4))
                 + y[l+96] * (s[6] - 32) * (((q[l] >> 6) & 3) - (h[l] & (m << 3) ? 0 : 4));
            sum += y[l+16] * (s[1] - 32) * (((q[l+16] >> 0) & 3) - (h[l+16] & (m << 0) ? 0 : 4))
                 + y[l+48] * (s[3] - 32) * (((q[l+16] >> 2) & 3) - (h[l+16] & (m << 1) ? 0 : 4))
                 + y[l+80] * (s[5] - 32) * (((q[l+16] >> 4) & 3) - (h[l+16] & (m << 2) ? 0 : 4))
                + y[l+112] * (s[7] - 32) * (((q[l+16] >> 6) & 3) - (h[l+16] & (m << 3) ? 0 : 4));
        }
        tmp += d * sum;

    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q4_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q4_K * x = (const block_q4_K *)vx + ib0;

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 8/K_QUANTS_PER_ITERATION;           // 8 or 4

    const int il  = tid/step;                            // 0...3
    const int ir  = tid - step*il;                       // 0...7 or 0...3
    const int n   = 2 * K_QUANTS_PER_ITERATION;          // 2 or 4

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

#if K_QUANTS_PER_ITERATION == 2
    uint32_t q32[4];
    const uint8_t * q4 = (const uint8_t *)q32;
#else
    uint16_t q16[4];
    const uint8_t * q4 = (const uint8_t *)q16;
#endif

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y1 = yy + i*QK_K + y_offset;
        const float   * y2 = y1 + 128;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

#if K_QUANTS_PER_ITERATION == 2
        const uint32_t * q1 = (const uint32_t *)(x[i].qs + q_offset);
        const uint32_t * q2 = q1 + 16;

        q32[0] = q1[0] & 0x0f0f0f0f;
        q32[1] = q1[0] & 0xf0f0f0f0;
        q32[2] = q2[0] & 0x0f0f0f0f;
        q32[3] = q2[0] & 0xf0f0f0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 4; ++l) {
            s.x += y1[l] * q4[l+0]; s.y += y1[l+32] * q4[l+ 4];
            s.z += y2[l] * q4[l+8]; s.w += y2[l+32] * q4[l+12];
            smin += y1[l] * sc[2] + y1[l+32] * sc[3] + y2[l] * sc[6] + y2[l+32] * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#else
        const uint16_t * q1 = (const uint16_t *)(x[i].qs + q_offset);
        const uint16_t * q2 = q1 + 32;

        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[0] & 0xf0f0;
        q16[2] = q2[0] & 0x0f0f;
        q16[3] = q2[0] & 0xf0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 2; ++l) {
            s.x += y1[l] * q4[l+0]; s.y += y1[l+32] * q4[l+2];
            s.z += y2[l] * q4[l+4]; s.w += y2[l+32] * q4[l+6];
            smin += y1[l] * sc[2] + y1[l+32] * sc[3] + y2[l] * sc[6] + y2[l+32] * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#endif

    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q5_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols) {

    const int row = blockIdx.x;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q5_K * x = (const block_q5_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/2;  // 0...15
    const int ix  = threadIdx.x%2;

    const int il  = tid/4;     // 0...3
    const int ir  = tid - 4*il;// 0...3
    const int n   = 2;

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    const uint8_t hm1  = 1 << (2*im);
    const uint8_t hm2  = hm1 << 4;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

    uint16_t q16[8];
    const uint8_t * q4 = (const uint8_t *)q16;

    for (int i = ix; i < num_blocks_per_row; i += 2) {

        const uint8_t * ql1 = x[i].qs + q_offset;
        const uint8_t * qh  = x[i].qh + l0;
        const float   * y1  = yy + i*QK_K + y_offset;
        const float   * y2  = y1 + 128;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

        float4 sum = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        const uint16_t * q1 = (const uint16_t *)ql1;
        const uint16_t * q2 = q1 + 32;
        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[8] & 0x0f0f;
        q16[2] = (q1[0] >> 4) & 0x0f0f;
        q16[3] = (q1[8] >> 4) & 0x0f0f;
        q16[4] = q2[0] & 0x0f0f;
        q16[5] = q2[8] & 0x0f0f;
        q16[6] = (q2[0] >> 4) & 0x0f0f;
        q16[7] = (q2[8] >> 4) & 0x0f0f;
        for (int l = 0; l < n; ++l) {
            sum.x += y1[l+ 0] * (q4[l +0] + (qh[l+ 0] & (hm1 << 0) ? 16 : 0))
                   + y1[l+16] * (q4[l +2] + (qh[l+16] & (hm1 << 0) ? 16 : 0));
            sum.y += y1[l+32] * (q4[l +4] + (qh[l+ 0] & (hm1 << 1) ? 16 : 0))
                   + y1[l+48] * (q4[l +6] + (qh[l+16] & (hm1 << 1) ? 16 : 0));
            sum.z += y2[l+ 0] * (q4[l +8] + (qh[l+ 0] & (hm2 << 0) ? 16 : 0))
                   + y2[l+16] * (q4[l+10] + (qh[l+16] & (hm2 << 0) ? 16 : 0));
            sum.w += y2[l+32] * (q4[l+12] + (qh[l+ 0] & (hm2 << 1) ? 16 : 0))
                   + y2[l+48] * (q4[l+14] + (qh[l+16] & (hm2 << 1) ? 16 : 0));
            smin += (y1[l] + y1[l+16]) * sc[2] + (y1[l+32] + y1[l+48]) * sc[3]
                  + (y2[l] + y2[l+16]) * sc[6] + (y2[l+32] + y2[l+48]) * sc[7];
        }
        tmp += dall * (sum.x * sc[0] + sum.y * sc[1] + sum.z * sc[4] + sum.w * sc[5]) - dmin * smin;
    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

static __global__ void dequantize_mul_mat_vec_q6_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q6_K * x = (const block_q6_K *)vx + ib0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0, 1

    const int step = 16/K_QUANTS_PER_ITERATION;          // 16 or 8

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

#if K_QUANTS_PER_ITERATION == 1
    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15
    const int is = 0;
#else
    const int l0 = 4 * in;                               // 0, 4, 8, ..., 28
    const int is = in / 4;
#endif
    const int ql_offset = 64*im + l0;
    const int qh_offset = 32*im + l0;
    const int s_offset  =  8*im + is;
    const int y_offset = 128*im + l0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * ql = x[i].ql + ql_offset;
        const uint8_t * qh = x[i].qh + qh_offset;
        const int8_t  * s  = x[i].scales + s_offset;

        const float d = x[i].d;

#if K_QUANTS_PER_ITERATION == 1
        float sum = y[ 0] * s[0] * d * ((int8_t)((ql[ 0] & 0xF) | ((qh[ 0] & 0x03) << 4)) - 32)
                  + y[16] * s[1] * d * ((int8_t)((ql[16] & 0xF) | ((qh[16] & 0x03) << 4)) - 32)
                  + y[32] * s[2] * d * ((int8_t)((ql[32] & 0xF) | ((qh[ 0] & 0x0c) << 2)) - 32)
                  + y[48] * s[3] * d * ((int8_t)((ql[48] & 0xF) | ((qh[16] & 0x0c) << 2)) - 32)
                  + y[64] * s[4] * d * ((int8_t)((ql[ 0]  >> 4) | ((qh[ 0] & 0x30) >> 0)) - 32)
                  + y[80] * s[5] * d * ((int8_t)((ql[16]  >> 4) | ((qh[16] & 0x30) >> 0)) - 32)
                  + y[96] * s[6] * d * ((int8_t)((ql[32]  >> 4) | ((qh[ 0] & 0xc0) >> 2)) - 32)
                  +y[112] * s[7] * d * ((int8_t)((ql[48]  >> 4) | ((qh[16] & 0xc0) >> 2)) - 32);
        tmp += sum;
#else
        float sum = 0;
        for (int l = 0; l < 4; ++l) {
            sum += y[l+ 0] * s[0] * d * ((int8_t)((ql[l+ 0] & 0xF) | (((qh[l] >> 0) & 3) << 4)) - 32)
                 + y[l+32] * s[2] * d * ((int8_t)((ql[l+32] & 0xF) | (((qh[l] >> 2) & 3) << 4)) - 32)
                 + y[l+64] * s[4] * d * ((int8_t)((ql[l+ 0]  >> 4) | (((qh[l] >> 4) & 3) << 4)) - 32)
                 + y[l+96] * s[6] * d * ((int8_t)((ql[l+32]  >> 4) | (((qh[l] >> 6) & 3) << 4)) - 32);
        }
        tmp += sum;
#endif

    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (tid == 0) {
        dst[row] = tmp;
    }
}

static __device__ void convert_f16(const void * vx, const int64_t ib, const int iqs, dfloat2 & v){
    const half * x = (const half *) vx;
    // load 2 halfs into register in a single instruction
    const half2 x_reg = *((half2 *) &(x[ib + iqs]));
    // automatic half -> float type cast if dfloat == float
    v.x = __low2float(x_reg);
    v.y = __high2float(x_reg);
}

static constexpr __device__ dequantize_kernel_t get_dequantize_kernel(ggml_type type) {
    return type == GGML_TYPE_Q4_0 ? dequantize_q4_0 :
        type == GGML_TYPE_Q4_1 ? dequantize_q4_1 :
        type == GGML_TYPE_Q5_0 ? dequantize_q5_0 :
        type == GGML_TYPE_Q5_1 ? dequantize_q5_1 :
        type == GGML_TYPE_Q8_0 ? dequantize_q8_0 :
        type == GGML_TYPE_F16 ? convert_f16 :
        nullptr;
}

template <ggml_type type>
static __global__ void dequantize_mul_mat_vec(const void * __restrict__ vx, const dfloat * __restrict__ y, float * __restrict__ dst, const int ncols, const int nrows) {
    constexpr int qk = ggml_cuda_type_traits<type>::qk; // quantized weights per x block
    constexpr int qr = ggml_cuda_type_traits<type>::qr; // number of quantized weights per data value in x block
    constexpr dequantize_kernel_t dequantize_kernel = get_dequantize_kernel(type);

    const int64_t row = (int64_t)blockIdx.x*blockDim.y + threadIdx.y;

    if (row >= nrows) {
        return;
    }

    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

// partial sum for each thread
#ifdef GGML_CUDA_F16
    half2 tmp = {0.0f, 0.0f}; // two sums for f16 to take advantage of half2 intrinsics
#else
    float tmp = 0.0f;
#endif // GGML_CUDA_F16

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int64_t ib = ((int64_t)row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val
            dfloat2 v;
            dequantize_kernel(vx, ib, iqs + j/qr, v);

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
#ifdef GGML_CUDA_F16
            if ( y_offset == 1 ) {
                // load 2 dfloats into register in a single instruction
                const dfloat2 y_reg = *((dfloat2 *) &(y[iybs + iqs + j/qr]));
                tmp += __hmul2(v, y_reg);
            }
            else {
                tmp += __hmul2(v, {
                        y[iybs + iqs + j/qr + 0],
                        y[iybs + iqs + j/qr + y_offset]
                    });
            }
#else
            if ( y_offset == 1 ) {
                // load 2 dfloats into register in a single instruction
                const dfloat2 y_reg = *((dfloat2 *) &(y[iybs + iqs + j/qr]));
                tmp += v.x * y_reg.x;
                tmp += v.y * y_reg.y;
            }
            else {
                tmp += v.x * y[iybs + iqs + j/qr + 0];
                tmp += v.y * y[iybs + iqs + j/qr + y_offset];
            }
#endif // GGML_CUDA_F16
        }
    }

    // sum up partial sums and write back result
    tmp = warp_reduce_sum(tmp);

    if (tid == 0) {
#ifdef GGML_CUDA_F16
        dst[row] = tmp.x + tmp.y;
#else
        dst[row] = tmp;
#endif // GGML_CUDA_F16
    }
}

static void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    // the number of rows may exceed maximum grid size in the y or z dimensions, use the x dimension instead
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_Q4_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_Q4_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_Q5_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_Q5_1>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_Q8_0>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q2_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2; // very slightly faster than 1 even when K_QUANTS_PER_ITERATION = 2
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q2_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q3_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q3_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q4_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q4_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void dequantize_mul_mat_vec_q5_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const dim3 block_dims(32, 1, 1);
    dequantize_mul_mat_vec_q5_k<<<nrows, block_dims, 0, stream>>>(vx, y, dst, ncols);
}

static void dequantize_mul_mat_vec_q6_K_cuda(const void * vx, const float * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % QK_K == 0);
    const int ny = 2 / K_QUANTS_PER_ITERATION;
    const int block_num_y = (nrows + ny - 1) / ny;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(32, ny, 1);
    dequantize_mul_mat_vec_q6_k<<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

static void convert_mul_mat_vec_f16_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows, hipStream_t stream) {
    GGML_ASSERT(ncols % (GGML_CUDA_DMMV_X*2) == 0);
    const int block_num_y = (nrows + GGML_CUDA_MMV_Y - 1) / GGML_CUDA_MMV_Y;
    const dim3 block_nums(block_num_y, 1, 1);
    const dim3 block_dims(WARP_SIZE, GGML_CUDA_MMV_Y, 1);
    dequantize_mul_mat_vec<GGML_TYPE_F16>
        <<<block_nums, block_dims, 0, stream>>>(vx, y, dst, ncols, nrows);
}

void ggml_cuda_op_dequantize_mul_mat_vec(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {
    GGML_UNUSED(ctx);
    const int64_t ne00 = src0->ne[0];
    const int64_t row_diff = row_high - row_low;

    GGML_ASSERT(src1->type == GGML_TYPE_F32);

    // on some GPUs it is faster to convert src1 to half and to use half precision intrinsics
#ifdef GGML_CUDA_F16
    ggml_cuda_pool_alloc<half> src1_dfloat_a(ctx.pool());
    half * src1_dfloat = nullptr; // dfloat == half

    bool src1_convert_f16 =
        src0->type == GGML_TYPE_Q4_0 || src0->type == GGML_TYPE_Q4_1 ||
        src0->type == GGML_TYPE_Q5_0 || src0->type == GGML_TYPE_Q5_1 ||
        src0->type == GGML_TYPE_Q8_0 || src0->type == GGML_TYPE_F16;

    if (src1_convert_f16) {
        src1_dfloat = src1_dfloat_a.alloc(ne00);
        const to_fp16_cuda_t to_fp16_cuda = ggml_get_to_fp16_cuda(src1->type);
        GGML_ASSERT(to_fp16_cuda != nullptr);
        to_fp16_cuda(src1_ddf_i, src1_dfloat, ne00, stream);
    }
#else
    const dfloat * src1_dfloat = (const dfloat *) src1_ddf_i; // dfloat == float, no conversion
#endif // GGML_CUDA_F16

    switch (src0->type) {
        case GGML_TYPE_Q4_0:
            dequantize_mul_mat_vec_q4_0_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q4_1:
            dequantize_mul_mat_vec_q4_1_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q5_0:
            dequantize_mul_mat_vec_q5_0_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q5_1:
            dequantize_mul_mat_vec_q5_1_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q8_0:
            dequantize_mul_mat_vec_q8_0_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q2_K:
            dequantize_mul_mat_vec_q2_K_cuda(src0_dd_i, src1_ddf_i, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q3_K:
            dequantize_mul_mat_vec_q3_K_cuda(src0_dd_i, src1_ddf_i, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q4_K:
            dequantize_mul_mat_vec_q4_K_cuda(src0_dd_i, src1_ddf_i, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q5_K:
            dequantize_mul_mat_vec_q5_K_cuda(src0_dd_i, src1_ddf_i, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_Q6_K:
            dequantize_mul_mat_vec_q6_K_cuda(src0_dd_i, src1_ddf_i, dst_dd_i, ne00, row_diff, stream);
            break;
        case GGML_TYPE_F16:
            convert_mul_mat_vec_f16_cuda(src0_dd_i, src1_dfloat, dst_dd_i, ne00, row_diff, stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }

    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddq_i);
    GGML_UNUSED(src1_ncols);
    GGML_UNUSED(src1_padded_row_size);
}

bool ggml_cuda_dmmv_type_supported(ggml_type src0_type) {
    return src0_type == GGML_TYPE_Q4_0 || src0_type == GGML_TYPE_Q4_1 ||
        src0_type == GGML_TYPE_Q5_0 || src0_type == GGML_TYPE_Q5_1 ||
        src0_type == GGML_TYPE_Q8_0 || src0_type == GGML_TYPE_Q2_K ||
        src0_type == GGML_TYPE_Q3_K || src0_type == GGML_TYPE_Q4_K ||
        src0_type == GGML_TYPE_Q5_K || src0_type == GGML_TYPE_Q6_K ||
        src0_type == GGML_TYPE_F16;
}
