#include "hip/hip_runtime.h"
#include "opt-step-adamw.cuh"

#include <cstdint>

static __global__ void opt_step_adamw_f32(
    float * __restrict__ x, const float * __restrict__ g, float * __restrict__ g_m, float * __restrict__ g_v, const int64_t k,
    const float alpha, const float beta1, const float beta2, const float eps, const float wd,
    const float beta1h, const float beta2h) {

    const int64_t i = (int64_t) blockIdx.x*blockDim.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const float gi = g[i];
    const float gmi = g_m[i]*beta1 +    gi*(1.0f - beta1);
    const float gvi = g_v[i]*beta2 + gi*gi*(1.0f - beta2);

    g_m[i] = gmi;
    g_v[i] = gvi;

    const float mh =       gmi*beta1h;
    const float vh = sqrtf(gvi*beta2h) + eps;

    x[i] = x[i]*(1.0f - alpha*wd) - mh/vh;
}

static void opt_step_adamw_f32_cuda(
    float * x, const float * g, float * g_m, float * g_v, const int64_t k,
    const float alpha, const float beta1, const float beta2, const float eps, const float wd,
    const float beta1h, const float beta2h, hipStream_t stream) {

    const dim3 block_dims(CUDA_OPT_STEP_ADAMW_BLOCK_SIZE, 1, 1);
    const dim3 block_nums((k + CUDA_OPT_STEP_ADAMW_BLOCK_SIZE - 1) / CUDA_OPT_STEP_ADAMW_BLOCK_SIZE, 1, 1);
    opt_step_adamw_f32<<<block_nums, block_dims, 0, stream>>>(x, g, g_m, g_v, k, alpha, beta1, beta2, eps, wd, beta1h, beta2h);
}

void ggml_cuda_opt_step_adamw(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0        = dst->src[0];
    const ggml_tensor * src0_grad   = dst->src[1];
    const ggml_tensor * src0_grad_m = dst->src[2];
    const ggml_tensor * src0_grad_v = dst->src[3];

    GGML_ASSERT(src0->type        == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad->type   == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad_m->type == GGML_TYPE_F32);
    GGML_ASSERT(src0_grad_v->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(src0_grad));
    GGML_ASSERT(ggml_is_contiguous(src0_grad_m));
    GGML_ASSERT(ggml_is_contiguous(src0_grad_v));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad_m));
    GGML_ASSERT(ggml_are_same_shape(src0, src0_grad_v));

    float       * src0_d        = (float       *) src0->data;
    const float * src0_grad_d   = (const float *) src0_grad->data;
    float       * src0_grad_m_d = (float       *) src0_grad_m->data;
    float       * src0_grad_v_d = (float       *) src0_grad_v->data;

    hipStream_t stream = ctx.stream();

    const int64_t ne = ggml_nelements(src0);

    int64_t iter;  memcpy(&iter,  &dst->op_params[0], sizeof(int64_t));
    float   alpha; memcpy(&alpha, &dst->op_params[2], sizeof(float));
    float   beta1; memcpy(&beta1, &dst->op_params[3], sizeof(float));
    float   beta2; memcpy(&beta2, &dst->op_params[4], sizeof(float));
    float   eps;   memcpy(&eps,   &dst->op_params[5], sizeof(float));
    float   wd;    memcpy(&wd,    &dst->op_params[6], sizeof(float));

    const float beta1h  = alpha/(1.0f - powf(beta1, iter));
    const float beta2h  =  1.0f/(1.0f - powf(beta2, iter));

    opt_step_adamw_f32_cuda(src0_d, src0_grad_d, src0_grad_m_d, src0_grad_v_d, ne, alpha, beta1, beta2, eps, wd, beta1h, beta2h, stream);

    iter++;
    memcpy(&dst->op_params[0], &iter, sizeof(int64_t));
}
